#include "hip/hip_runtime.h"

#include "Random.cuh"

#include "Scene.h"

KERNEL void KrnlSetupRNG(CScene* pDevScene, hiprandStateXORWOW_t* pDevRandomStates)
{
	const int X		= (blockIdx.x * blockDim.x) + threadIdx.x;
	const int Y		= (blockIdx.y * blockDim.y) + threadIdx.y;

	// Exit if beyond canvas boundaries
	if (X >= pDevScene->m_Camera.m_Film.m_Resolution.GetResX() || Y >= pDevScene->m_Camera.m_Film.m_Resolution.GetResY())
		return;

	// Initialize
	hiprand_init(Y * (int)pDevScene->m_Camera.m_Film.m_Resolution.GetResX() + X, 1234, 0, &pDevRandomStates[Y * (int)pDevScene->m_Camera.m_Film.m_Resolution.GetResY() + X]);
}

extern "C" void SetupRNG(CScene* pScene, CScene* pDevScene, hiprandStateXORWOW_t* pDevRandomStates)
{
	const dim3 KernelBlock(32, 8);
	const dim3 KernelGrid((int)ceilf((float)pScene->m_Camera.m_Film.m_Resolution.GetResX() / (float)KernelBlock.x), (int)ceilf((float)pScene->m_Camera.m_Film.m_Resolution.GetResY() / (float)KernelBlock.y));

	KrnlSetupRNG<<<KernelGrid, KernelBlock>>>(pDevScene, pDevRandomStates);

	hipError_t Error = hipGetLastError();
}