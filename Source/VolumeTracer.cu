#include "hip/hip_runtime.h"

#include "VolumeTracer.cuh"

#include "Filter.h"
#include "Scene.h"
#include "Material.h"

texture<short, 3, hipReadModeNormalizedFloat>	gTexDensity;

hipArray* gpI = NULL;

KERNEL void KrnlSetupRNG(CScene* pDevScene, hiprandStateXORWOW_t* pDevRandomStates)
{
	const int X		= (blockIdx.x * blockDim.x) + threadIdx.x;
	const int Y		= (blockIdx.y * blockDim.y) + threadIdx.y;

	// Exit if beyond canvas boundaries
	if (X >= pDevScene->m_Camera.m_Film.m_Resolution.Width() || Y >= pDevScene->m_Camera.m_Film.m_Resolution.Height())
		return;

	// Initialize
	hiprand_init(Y * (int)pDevScene->m_Camera.m_Film.m_Resolution.Width() + X, 1234, 0, &pDevRandomStates[Y * (int)pDevScene->m_Camera.m_Film.m_Resolution.Width() + X]);
}

extern "C" void SetupRNG(CScene* pScene, CScene* pDevScene, hiprandStateXORWOW_t* pDevRandomStates)
{
	const dim3 KernelBlock(32, 8);
	const dim3 KernelGrid((int)ceilf((float)pScene->m_Camera.m_Film.m_Resolution.Width() / (float)KernelBlock.x), (int)ceilf((float)pScene->m_Camera.m_Film.m_Resolution.Height() / (float)KernelBlock.y));

	KrnlSetupRNG<<<KernelGrid, KernelBlock>>>(pDevScene, pDevRandomStates);

	hipError_t Error = hipGetLastError();
}

void BindVolumeData(short* pDensity, CResolution3D& Resolution)
{
	hipExtent ExtentGridI;

	ExtentGridI.width	= Resolution.m_XYZ.x;
	ExtentGridI.depth	= Resolution.m_XYZ.z;
	ExtentGridI.height	= Resolution.m_XYZ.y; 

	// create 3D array
	hipChannelFormatDesc ChannelDescDensity = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpI, &ChannelDescDensity, ExtentGridI);

	// copy data to 3D array
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr(pDensity, ExtentGridI.width * sizeof(short), ExtentGridI.width, ExtentGridI.height);
	copyParams.dstArray = gpI;
	copyParams.extent   = ExtentGridI;
	copyParams.kind     = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, gpI, ChannelDescDensity);
}

void UnbindVolumeData(void)
{
	hipFree(gpI);
}

DEV float Density(const Vec3f& P)
{
	return (float)(SHRT_MAX * tex3D(gTexDensity, P.x, P.y, P.z));
}

// Computes the attenuation through the volume
DEV CColorXyz Transmittance(CScene* pDevScene, const Vec3f& P, const Vec3f& D, const float& MaxT, const float& StepSize, CCudaRNG& RNG)
{
	// Near and far intersections with volume axis aligned bounding box
	float NearT = 0.0f, FarT = FLT_MAX;

	if (!pDevScene->m_BoundingBox.Intersect(CRay(P, D, 0.0f, MaxT), &NearT, &FarT))
		return SPEC_WHITE;

	CColorXyz Lt = SPEC_WHITE;

	NearT += RNG.Get1() * StepSize;

	// Accumulate
	while (NearT < MaxT)
	{
		// Determine sample point
		const Vec3f SP = P + D * (NearT);

		// Fetch density
		const float D = Density(SP);
		
		// We ignore air density
		if (D == 0)
		{
			// Increase extent
			NearT += StepSize;
			continue;
		}

		// Get shadow opacity
		const float		Opacity = pDevScene->m_TransferFunctions.m_Opacity.F(D).r;
		const CColorXyz	Color	= pDevScene->m_TransferFunctions.m_DiffuseColor.F(D).ToXYZ();

		if (Opacity > 0.0f)
		{
			// Compute chromatic attenuation
			Lt.c[0] *= expf(-(Opacity * (1.0f - Color.c[0]) * StepSize));
			Lt.c[1] *= expf(-(Opacity * (1.0f - Color.c[1]) * StepSize));
			Lt.c[2] *= expf(-(Opacity * (1.0f - Color.c[2]) * StepSize));

			// Exit if eye transmittance is very small
			if (Lt.y() < 0.05f)
				break;
		}

		// Increase extent
		NearT += StepSize;
	}

	return Lt;
}

// Estimates direct lighting
DEV CColorXyz EstimateDirectLight(CScene* pDevScene, CLight& Light, CLightingSample& LS, const Vec3f& Wo, const Vec3f& Pe, const Vec3f& N, CCudaRNG& Rnd, const float& StepSize)
{
	
	if (Dot(Wo, N) < 0.0f)
		return SPEC_BLACK;

	// Accumulated radiance
	CColorXyz Ld = SPEC_BLACK;
	
	// Radiance from light source
	CColorXyz Li = SPEC_BLACK;

	// Attenuation
	CColorXyz Tr = SPEC_BLACK;

	float D = Density(Pe);

	CBSDF Bsdf(N, Wo, pDevScene->m_TransferFunctions.m_DiffuseColor.F(D).ToXYZ(), pDevScene->m_TransferFunctions.m_SpecularColor.F(D).ToXYZ(), 1.0f, 1.0f);
	// Light/shadow ray
	CRay R; 

	// Light probability
	float LightPdf = 1.0f, BsdfPdf = 1.0f;
	
	// Incident light direction
	Vec3f Wi;

	CColorXyz F = SPEC_BLACK;
	
	CSurfacePoint SPe, SPl;

	SPe.m_P		= Pe;
	SPe.m_Ng	= N; 

	// Sample the light source
 	Li = Light.SampleL(SPe, SPl, LS, LightPdf, 0.1f);
	
	R.m_O		= SPl.m_P;
	R.m_D		= Normalize(SPe.m_P - SPl.m_P);
	R.m_MinT	= 0.0f;
	R.m_MaxT	= (SPl.m_P - SPe.m_P).Length();
	
	Wi = -R.m_D; 

	F = Bsdf.F(Wo, Wi); 

	BsdfPdf	= Bsdf.Pdf(Wo, Wi);
//	BsdfPdf = Dot(Wi, N);

	// Sample the light with MIS
	if (!Li.IsBlack() && LightPdf > 0.0f && BsdfPdf > 0.0f)
	{
		// Compute tau
		const CColorXyz Tr = Transmittance(pDevScene, R.m_O, R.m_D, Length(R.m_O - Pe), StepSize, Rnd);
		
		// Attenuation due to volume
		Li *= Tr;

		// Compute MIS weight
		const float Weight = 1.0f;//PowerHeuristic(1.0f, LightPdf, 1.0f, BsdfPdf);
 
		// Add contribution
		Ld += F * Li * (AbsDot(Wi, N) * Weight / LightPdf);
	}
	/**/

	/*
	// Sample the BRDF with MIS
	F = Bsdf.SampleF(Wo, Wi, BsdfPdf, LS.m_BsdfSample);
	
//	Wi = CosineWeightedHemisphere(Rnd.Get2(), N);

//	BsdfPdf = Dot(Wi, N);

	CLight* pNearestLight = NULL;

	Vec2f UV;

	if (!F.IsBlack())
	{
		float MaxT = INF_MAX;

		// Compute virtual light point
		const Vec3f Pl = Pe + (MaxT * Wi);

		if (NearestLight(pScene, Pe, Wi, 0.0f, MaxT, pNearestLight, NULL, &UV, &LightPdf))
		{
			if (LightPdf > 0.0f && BsdfPdf > 0.0f) 
			{
				// Add light contribution from BSDF sampling
				const float Weight = PowerHeuristic(1.0f, BsdfPdf, 1.0f, LightPdf);
				 
				// Get exitant radiance from light source
				Li = pNearestLight->Le(UV, pScene->m_Materials, pScene->m_Textures, pScene->m_Bitmaps);

				if (!Li.IsBlack())
				{
					// Scale incident radiance by attenuation through volume
					Tr = Transmittance(pScene, Pe, Wi, 1.0f, StepSize, Rnd);

					// Attenuation due to volume
					Li *= Tr;

					// Contribute
					Ld += F * Li * AbsDot(Wi, N) * Weight / BsdfPdf;
				}
			}
		}
	}
	*/

	return SPEC_WHITE;
}

// Uniformly samples one light
DEV CColorXyz UniformSampleOneLight(CScene* pDevScene, const Vec3f& Wo, const Vec3f& Pe, const Vec3f& N, CCudaRNG& Rnd, const float& StepSize)
{
 	if (pDevScene->m_Lighting.m_NoLights == 0)
 		return SPEC_BLACK;

	CLightingSample LS;

	// Create light sampler
	LS.LargeStep(Rnd);

	// Choose which light to sample
	const int WhichLight = (int)floorf(LS.m_LightNum * (float)pDevScene->m_Lighting.m_NoLights);

	// Get the light
	CLight& Light = pDevScene->m_Lighting.m_Lights[WhichLight];

	// Return estimated direct light
	return (float)pDevScene->m_Lighting.m_NoLights * EstimateDirectLight(pDevScene, Light, LS, Wo, Pe, N, Rnd, StepSize);
}

HOD float PhaseHG(const Vec3f& W, const Vec3f& Wp, float G)
{
	float CosTheta = Dot(W, Wp);
	return 1.0f / (4.0f * PI_F) * (1.0f - G * G) / powf(1.0f + G * G - 2.0f * G * CosTheta, 1.5f);
}

HOD Vec3f SampleHG(const Vec3f& W, float G, const Vec2f& U)
{
	float CosTheta;

	if (fabsf(G) < 1e-3)
	{
		CosTheta = 1.0f - 2.0f * U.x;
	}
	else
	{
		float SqrtTerm = (1.0f - G * G) / (1.0f - G + 2.0f * G * U.x);
		CosTheta = (1.0f + G * G - SqrtTerm * SqrtTerm) / (2.0f * G);
	}

	float SinTheta = sqrtf(max(0.f, 1.f - CosTheta * CosTheta));
	float Phi = 2.f * PI_F * U.y;
	Vec3f V1, V2;
	CoordinateSystem(W, &V1, &V2);
	return SphericalDirection(SinTheta, CosTheta, Phi, V1, V2, W);
}

HOD float PdfHG(const Vec3f& W, const Vec3f& Wp, float G)
{
	return PhaseHG(W, Wp, G);
}

// Fetches the density from the texture
DEV inline float LookupDensity(const Vec3f& P)
{
	return (float)(SHRT_MAX * tex3D(gTexDensity, P.x, P.y, P.z));
}

// Computes the local gradient
DEV Vec3f ComputeGradient(const Vec3f& P)
{
	Vec3f Normal;

	Vec3f X(1.0f, 0.0f, 0.0f), Y(0.0f, 1.0f, 0.0f), Z(0.0f, 0.0f, 1.0f);

	Normal.x = 0.5f * (float)(Density(P + X) - Density(P - X));
	Normal.y = 0.5f * (float)(Density(P + Y) - Density(P - Y));
	Normal.z = 0.5f * (float)(Density(P + Z) - Density(P - Z));

	return -Normal;
}

DEV inline bool SampleDistanceRM(CRay& R, CCudaRNG& RNG, CVolumePoint& VP, CScene* pDevScene, int Component)
{
	float MinT = 0.0f, MaxT = 0.0f;

	if (!pDevScene->m_BoundingBox.Intersect(R, &MinT, &MaxT))
		return false;

	MinT = max(MinT, R.m_MinT);
	MaxT = min(MaxT, R.m_MaxT);

	float S = -log(RNG.Get1()) / pDevScene->m_MaxD, Dt = 1.0f * (1.0f / (float)pDevScene->m_Resolution.m_XYZ.Max()), Sum = 0.0f, SigmaT = 0.0f, D = 0.0f;

	Vec3f samplePos; 

	MinT += RNG.Get1() * Dt;

	while (Sum < S)
	{
		samplePos = R.m_O + MinT * R.m_D;

		if (MinT > MaxT)
			return false;
		
		D = (float)(SHRT_MAX * tex3D(gTexDensity, pDevScene->m_BoundingBox.m_MinP.x + (samplePos.x / pDevScene->m_BoundingBox.m_MaxP.x), pDevScene->m_BoundingBox.m_MinP.y + (samplePos.y / pDevScene->m_BoundingBox.m_MaxP.y), pDevScene->m_BoundingBox.m_MinP.z + (samplePos.z / pDevScene->m_BoundingBox.m_MaxP.z)));

		SigmaT	= 10.0f * pDevScene->m_TransferFunctions.m_Opacity.F(D)[Component] * pDevScene->m_TransferFunctions.m_DiffuseColor.F(D)[Component];
		Sum		+= SigmaT * Dt;
		MinT	+= Dt;
	}

	VP.m_Transmittance.c[Component]	= 0.5f;
	VP.m_P							= samplePos;
	VP.m_D							= D;

	return true;
}

DEV inline bool FreePathRM(CRay& R, CCudaRNG& RNG, CVolumePoint& VP, CScene* pDevScene, int Component)
{
	float MinT = 0.0f, MaxT = 0.0f;

	if (!pDevScene->m_BoundingBox.Intersect(R, &MinT, &MaxT))
		return false;

	MinT = max(MinT, R.m_MinT);
//	MaxT = min(MaxT, R.m_MaxT);

	float S = -log(RNG.Get1()) / pDevScene->m_MaxD, Dt = 1.0f * (1.0f / (float)pDevScene->m_Resolution.m_XYZ.Max()), Sum = 0.0f, SigmaT = 0.0f, D = 0.0f;

	Vec3f samplePos; 

	MinT += RNG.Get1() * Dt;

	while (Sum < S)
	{
		samplePos = R.m_O + MinT * R.m_D;

		// Free path, no collisions in between
		if (MinT > R.m_MaxT)
			break;
		
		D = (float)(SHRT_MAX * tex3D(gTexDensity, pDevScene->m_BoundingBox.m_MinP.x + (samplePos.x / pDevScene->m_BoundingBox.m_MaxP.x), pDevScene->m_BoundingBox.m_MinP.y + (samplePos.y / pDevScene->m_BoundingBox.m_MaxP.y), pDevScene->m_BoundingBox.m_MinP.z + (samplePos.z / pDevScene->m_BoundingBox.m_MaxP.z)));

		SigmaT	= 10.0f * pDevScene->m_TransferFunctions.m_Opacity.F(D)[Component] * pDevScene->m_TransferFunctions.m_DiffuseColor.F(D)[Component];
		Sum		+= SigmaT * Dt;
		MinT	+= Dt;
	}

	if (MinT < R.m_MaxT)
		return false;

	VP.m_Transmittance.c[Component]	= 0.5f;
	VP.m_P							= samplePos;
	VP.m_D							= D;

	return true;
}

// Trace volume with single scattering
KERNEL void KrnlRenderVolume(CScene* pDevScene, hiprandStateXORWOW_t* pDevRandomStates, CColorXyz* pDevEstFrameXyz)
{
	const int X = (blockIdx.x * blockDim.x) + threadIdx.x;		// Get global y
	const int Y	= (blockIdx.y * blockDim.y) + threadIdx.y;		// Get global x

	// Compute sample ID
	const int SID = (Y * (gridDim.x * blockDim.x)) + X;

	// Exit if beyond kernel boundaries
	if (X >= pDevScene->m_Camera.m_Film.m_Resolution.Width() || Y >= pDevScene->m_Camera.m_Film.m_Resolution.Height())
		return;

	// Init random number generator
	CCudaRNG RNG(&pDevRandomStates[SID]);

	CRay Re, Rl;
	
	// Generate the camera ray
	pDevScene->m_Camera.GenerateRay(Vec2f(X, Y), RNG.Get2(), Re.m_O, Re.m_D);

	// Eye attenuation (Le), accumulated color through volume (Lv), unattenuated light from light source (Li), attenuated light from light source (Ld), and BSDF value (F)
	CColorXyz PathThroughput	= SPEC_WHITE;
	CColorXyz Li				= SPEC_BLACK;
	CColorXyz Lv				= SPEC_BLACK;
	CColorXyz F					= SPEC_BLACK;

	int NoScatteringEvents = 0, RussianRouletteDepth = 2; 

	Re.m_MinT	= 0.0f;
	Re.m_MaxT	= RAY_MAX;

	// Continue probability (Pc) Light probability (LightPdf) Bsdf probability (BsdfPdf)
	float Pc = 0.5f, LightPdf = 1.0f, BsdfPdf = 1.0f;

	// Eye point (Pe), light sample point (Pl), Gradient (G), normalized gradient (Gn), reversed eye direction (Wo), incident direction (Wi), new direction (W)
	Vec3f Pe, Pl, G, Gn, Wo, Wi, W;

	// Choose color component to sample
	int CC1 = floorf(RNG.Get1() * 3.0f);

	// Walk along the eye ray with ray marching
	while (NoScatteringEvents < pDevScene->m_MaxNoBounces)
	{
		CVolumePoint VP;

		// Sample distance
		if (SampleDistanceRM(Re, RNG, VP, pDevScene, CC1))
		{
// 			if (VP.m_Transmittance.y() > 0.0f)
//			PathThroughput.c[CC1] *= VP.m_Transmittance.c[CC1];

			// Compute gradient (G) and normalized gradient (Gn)
  			G	= ComputeGradient(VP.m_P);
  			Gn	= Normalize(G);
 			Wo	= Normalize(-Re.m_D);

			// Choose random light and compute the amount of light that reaches the scattering point
//			Li = SampleRandomLight(pScene, RNG, Pe, Pl, LightPdf);
//			Li = 1000.0f * CColorXyz(0.9f, 0.6f, 0.2f);
			Li = 500.0f * CColorXyz(1.0f);

			Pe = VP.m_P;



//			Pl = pDevScene->m_BoundingBox.GetCenter() + pDevScene->m_Light.m_Distance * Vec3f(sinf(pDevScene->m_Light.m_Theta), sinf(pDevScene->m_Light.m_Phi), cosf(pDevScene->m_Light.m_Theta));
//			Pl = pBoundingBox->GetCenter() + UniformSampleSphere(RNG.Get2()) * Vec3f(1000.0f);

			// LightPdf = powf((Pe - Pl).Length(), 2.0f);

			Rl = CRay(Pl, Normalize(Pe - Pl), 0.0f, (Pe - Pl).Length());

			if (!Li.IsBlack() && LightPdf > 0.0f && FreePathRM(Rl, RNG, VP, pDevScene, CC1))
			{
				Li /= LightPdf;
				Lv.c[CC1] += PathThroughput.c[CC1] * Li.c[CC1] * PhaseHG(Wo, Rl.m_D, pDevScene->m_PhaseG);// * VP.m_Transmittance.c[CC1];// * ;
			}

			W = Normalize(SampleHG(Wo, pDevScene->m_PhaseG, RNG.Get2()));
//			W = UniformSampleSphere(RNG.Get2());
//			W = UniformSampleHemisphere(RNG.Get2(), Gn);

			// Configure eye ray
			Re = CRay(VP.m_P, W, 0.0f, RAY_MAX);

			// Russian roulette
			if (NoScatteringEvents >= RussianRouletteDepth)
			{
				if (RNG.Get1() > Pc)
					break;
				else
					PathThroughput.c[CC1] /= Pc;
			}

//			PathThroughput.c[CC1] /= 4.0f * PI_F;
//			PathThroughput.c[CC1] /= PhaseHG(Wo, Rl.m_D, PhaseG);

			// Add scattering event
			NoScatteringEvents++;
		}
		else
		{
			break;
		}
	}

//  	if (pBoundingBox->Intersect(Re))
//  		Lv += SPEC_WHITE;


	pDevEstFrameXyz[Y * (int)pDevScene->m_Camera.m_Film.m_Resolution.Width() + X].c[CC1] = Lv.c[CC1] / fmaxf(1.0f, NoScatteringEvents);
}

// Computes the attenuation through the volume
DEV inline CColorXyz TransmittanceRM(CScene* pDevScene, CCudaRNG& RNG, CRay& R,const float& StepSize)
{
	// Near and far intersections with volume axis aligned bounding box
	float NearT = 0.0f, FarT = 0.0f;

	// Intersect with volume axis aligned bounding box
	if (!pDevScene->m_BoundingBox.Intersect(R, &NearT, &FarT))
		return SPEC_BLACK;

	// Clamp to near plane if necessary
	if (NearT < 0.0f) 
		NearT = 0.0f;     

	CColorXyz Lt = SPEC_WHITE;

	NearT += RNG.Get1() * StepSize;

	// Accumulate
	while (NearT < R.m_MaxT)
	{
		// Determine sample point
		const Vec3f SP = R(NearT);

		// Fetch density
		const short D = (float)(SHRT_MAX * tex3D(gTexDensity, pDevScene->m_BoundingBox.m_MinP.x + (SP.x / pDevScene->m_BoundingBox.m_MaxP.x), pDevScene->m_BoundingBox.m_MinP.y + (SP.y / pDevScene->m_BoundingBox.m_MaxP.y), pDevScene->m_BoundingBox.m_MinP.z + (SP.z / pDevScene->m_BoundingBox.m_MaxP.z)));
		
		// We ignore air density
		if (D == 0)
		{
			// Increase extent
			NearT += StepSize;
			continue;
		}

		// Get shadow opacity
		const float	Opacity = pDevScene->m_TransferFunctions.m_Opacity.F(D).r;

		if (Opacity > 0.0f)
		{
			// Compute eye transmittance
			Lt *= expf(-(Opacity * StepSize));

			// Exit if eye transmittance is very small
			if (Lt.y() < 0.1f)
				break;
		}

		// Increase extent
		NearT += StepSize;
	}

	return Lt;
}


// Trace volume with single scattering
KERNEL void KrnlSS(CScene* pDevScene, hiprandStateXORWOW_t* pDevRandomStates, CColorXyz* pDevEstFrameXyz)
{
	const int X = (blockIdx.x * blockDim.x) + threadIdx.x;		// Get global y
	const int Y	= (blockIdx.y * blockDim.y) + threadIdx.y;		// Get global x

	// Compute sample ID
	const int SID = (Y * (gridDim.x * blockDim.x)) + X;

	// Exit if beyond kernel boundaries
	if (X >= pDevScene->m_Camera.m_Film.m_Resolution.Width() || Y >= pDevScene->m_Camera.m_Film.m_Resolution.Height())
		return;

	// Init random number generator
	CCudaRNG RNG(&pDevRandomStates[SID]);

	// Eye ray (Re), Light ray (Rl)
	CRay Re, Rl;

	// Generate the camera ray
	pDevScene->m_Camera.GenerateRay(Vec2f(X, Y), RNG.Get2(), Re.m_O, Re.m_D);

	// Distance towards nearest intersection with bounding box (MinT), distance to furthest intersection with bounding box (MaxT)
	float MinT = 0.0f, MaxT = INF_MAX;

	// Early ray termination if ray does not intersect with bounding box
	if (!pDevScene->m_BoundingBox.Intersect(Re, &MinT, &MaxT))
	{
		pDevEstFrameXyz[Y * (int)pDevScene->m_Camera.m_Film.m_Resolution.Width() + X] = SPEC_BLACK;
		return;
	}

	// Eye attenuation (Le), accumulated color through volume (Lv), unattenuated light from light source (Li), attenuated light from light source (Ld), and BSDF value (F)
	CColorXyz Ltr	= SPEC_WHITE;
	CColorXyz Li	= SPEC_BLACK;
	CColorXyz Lv	= SPEC_BLACK;
	CColorXyz F		= SPEC_BLACK;

	int NoScatteringEvents = 0, RussianRouletteDepth = 2; 

	Re.m_MinT	= 0.0f;
	Re.m_MaxT	= RAY_MAX;

	// Continue probability (Pc) Light probability (LightPdf) Bsdf probability (BsdfPdf)
	float Pc = 0.5f, LightPdf = 1.0f, BsdfPdf = 1.0f;

	// Eye point (Pe), light sample point (Pl), Gradient (G), normalized gradient (Gn), reversed eye direction (Wo), incident direction (Wi), new direction (W)
	Vec3f Pe, Pl, G, Gn, Wo, Wi, W;

	// Distance along eye ray (Te), step size (Ss), density (D)
	float Ss = 0.2f, Te = MinT + RNG.Get1() * Ss, D = 0.0f;

	// Choose color component to sample
	int CC1 = floorf(RNG.Get1() * 3.0f);

	bool Hit = false;

	// Walk along the eye ray with ray marching
	while (Te < MaxT)
	{
		// Determine new point on eye ray
		Pe = Re(Te);

		// Increase parametric range
		Te += Ss;

		// Fetch density
		const short D = Density(Pe);

		// We ignore air density
		if (D == 0.0f)
			continue;

		// Get opacity at eye point
		const float Tr = pDevScene->m_TransferFunctions.m_Opacity.F(D).r;
//		const CColorXyz	Ke = pDevScene->m_Volume.Ke(D);
		
		// Add emission
//		Ltr += Ke;

		// Compute outgoing direction
		Wo = Normalize(-Re.m_D);

		// Obtain normal
		Gn = ComputeGradient(Pe);

		// Exit if air, or not within hemisphere
		if (Tr < 0.01f)
			continue;

		// Estimate direct light at eye point
	 	Lv += Ltr * UniformSampleOneLight(pDevScene, Wo, Pe, Gn, RNG, Ss);
//		Lv += Ltr * T * SPEC_WHITE;

		// Compute eye transmittance
		Ltr *= expf(-(Tr * Ss));

		// Exit if eye transmittance is very small
// 		if (EyeTr.y() < gScene.m_Volume.m_TauThreshold)
// 			break;

		if (Ltr.y() < 0.1f)
		{
// 			EyeTr = 
			break;
		}
	}

	pDevEstFrameXyz[Y * (int)pDevScene->m_Camera.m_Film.m_Resolution.Width() + X] = Lv;
}


































// Traces the volume
void RenderVolume(CScene* pScene, CScene* pDevScene, hiprandStateXORWOW_t* pDevRandomStates, CColorXyz* pDevEstFrameXyz)
{
	const dim3 KernelBlock(pScene->m_KernelSize.x, pScene->m_KernelSize.y);
	const dim3 KernelGrid((int)ceilf((float)pScene->m_Camera.m_Film.m_Resolution.Width() / (float)KernelBlock.x), (int)ceilf((float)pScene->m_Camera.m_Film.m_Resolution.Height() / (float)KernelBlock.y));
	
	// Execute kernel
//	KrnlRenderVolume<<<KernelGrid, KernelBlock>>>(pDevScene, pDevRandomStates, pDevEstFrameXyz);
	KrnlSS<<<KernelGrid, KernelBlock>>>(pDevScene, pDevRandomStates, pDevEstFrameXyz);
}

KERNEL void KrnlBlurXyzH(CColorXyz* pImage, CColorXyz* pTempImage, CResolution2D Resolution, CGaussianFilter GaussianFilter)
{
	const int X 	= (blockIdx.x * blockDim.x) + threadIdx.x;		// Get global y
	const int Y		= (blockIdx.y * blockDim.y) + threadIdx.y;		// Get global x
	const int PID	= (Y * Resolution.m_XY.x) + X;					// Get pixel ID	

	// Exit if beyond image boundaries
	if (X >= Resolution.m_XY.x || Y >= Resolution.m_XY.y)
		return;

	// Compute filter extent
	const int X0 = max((int)ceilf(X - GaussianFilter.xWidth), 0);
	const int X1 = min((int)floorf(X + GaussianFilter.xWidth), Resolution.m_XY.x - 1);

	// Accumulated color
	CColorXyz Sum;

	// Weights
	float FW = 1.0f, SumW = 0.0f;

	for (int x = X0; x <= X1; x++)
	{
		// Compute filter weight
		FW = GaussianFilter.Evaluate(fabs((float)(x - X) / (0.5f * GaussianFilter.xWidth)), 0.0f);

		Sum		+= FW * pImage[(Y * Resolution.m_XY.x) + x];
		SumW	+= FW;
	}

	__syncthreads();

	// Write to temporary image
	pTempImage[PID] = Sum / SumW;
}

// ToDo: Add description
KERNEL void KrnlBlurXyzV(CColorXyz* pImage, CColorXyz* pTempImage, CResolution2D Resolution, CGaussianFilter GaussianFilter)
{
	const int X 	= (blockIdx.x * blockDim.x) + threadIdx.x;		// Get global y
	const int Y		= (blockIdx.y * blockDim.y) + threadIdx.y;		// Get global x
	const int PID	= (Y * Resolution.m_XY.x) + X;					// Get pixel ID	

	// Exit if beyond image boundaries
	if (X >= Resolution.m_XY.x || Y >= Resolution.m_XY.y)
		return;

	// Compute filter extent
	const int Y0 = max((int)ceilf (Y - GaussianFilter.yWidth), 0);
	const int Y1 = min((int)floorf(Y + GaussianFilter.yWidth), Resolution.m_XY.y - 1);

	// Accumulated color
	CColorXyz Sum;

	// Weights
	float FW = 1.0f, SumW = 0.0f;

	for (int y = Y0; y <= Y1; y++)
	{
		// Compute filter weight
		FW = GaussianFilter.Evaluate(0.0f, fabs((float)(y - Y) / (0.5f * GaussianFilter.yWidth)));

		Sum		+= FW * pTempImage[(y * Resolution.m_XY.x) + X];
		SumW	+= FW;
	}

	__syncthreads();

	// Write to image
	pImage[PID]	= Sum / SumW;
}

// ToDo: Add description
void BlurImageXyz(CColorXyz* pImage, CColorXyz* pTempImage, const CResolution2D& Resolution, const float& Radius)
{
	const dim3 KernelBlock(32, 8);
	const dim3 KernelGrid((int)ceilf((float)Resolution.m_XY.x / (float)KernelBlock.x), (int)ceilf((float)Resolution.m_XY.y / (float)KernelBlock.y));

	// Create gaussian filter
	CGaussianFilter GaussianFilter(2.0f * Radius, 2.0f * Radius, 2.0f);

	KrnlBlurXyzH<<<KernelGrid, KernelBlock>>>(pImage, pTempImage, Resolution, GaussianFilter); 
	KrnlBlurXyzV<<<KernelGrid, KernelBlock>>>(pImage, pTempImage, Resolution, GaussianFilter); 
}

// ToDo: Add description
KERNEL void KrnlComputeEstimate(int Width, int Height, CColorXyz* gpEstFrameXyz, CColorXyz* pAccEstXyz, float N, float Exposure, unsigned char* pPixels)
{
	const int X 	= (blockIdx.x * blockDim.x) + threadIdx.x;		// Get global Y
	const int Y		= (blockIdx.y * blockDim.y) + threadIdx.y;		// Get global X
	const int PID	= (Y * Width) + X;								// Get pixel ID	

	// Exit if beyond image boundaries
	if (X >= Width || Y >= Height)
		return;

	pAccEstXyz[PID] += gpEstFrameXyz[PID];

	const CColorXyz L = pAccEstXyz[PID] / (float)__max(1.0f, N);

	float InvGamma = 1.0f / 2.2f;

	CColorRgbHdr RgbHdr = CColorRgbHdr(L.c[0], L.c[1], L.c[2]);

	RgbHdr.r = Clamp(1.0f - expf(-(RgbHdr.r / Exposure)), 0.0, 1.0f);
	RgbHdr.g = Clamp(1.0f - expf(-(RgbHdr.g / Exposure)), 0.0, 1.0f);
	RgbHdr.b = Clamp(1.0f - expf(-(RgbHdr.b / Exposure)), 0.0, 1.0f);

	pPixels[(3 * (Y * Width + X)) + 0] = (unsigned char)Clamp((255.0f * powf(RgbHdr.r, InvGamma)), 0.0f, 255.0f);
	pPixels[(3 * (Y * Width + X)) + 1] = (unsigned char)Clamp((255.0f * powf(RgbHdr.g, InvGamma)), 0.0f, 255.0f);
	pPixels[(3 * (Y * Width + X)) + 2] = (unsigned char)Clamp((255.0f * powf(RgbHdr.b, InvGamma)), 0.0f, 255.0f);
}

void ComputeEstimate(int Width, int Height, CColorXyz* pEstFrameXyz, CColorXyz* pAccEstXyz, float N, float Exposure, unsigned char* pPixels)
{
	const dim3 KernelBlock(8, 8);
	const dim3 KernelGrid((int)ceilf((float)Width / (float)KernelBlock.x), (int)ceilf((float)Height / (float)KernelBlock.y));

	KrnlComputeEstimate<<<KernelGrid, KernelBlock>>>(Width, Height, pEstFrameXyz, pAccEstXyz, N, Exposure, pPixels); 
}