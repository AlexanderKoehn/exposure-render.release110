#include "hip/hip_runtime.h"

#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, 3, hipReadModeElementType>			gTexExtinction;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexEstimateRgbLdr;
texture<float, 1, hipReadModeElementType>			gTexOpacity;
texture<float4, 1, hipReadModeElementType>			gTexDiffuse;
texture<float4, 1, hipReadModeElementType>			gTexSpecular;
texture<float, 1, hipReadModeElementType>			gTexRoughness;
texture<float4, 1, hipReadModeElementType>			gTexEmission;

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
hipArray* gpOpacityArray				= NULL;
hipArray* gpDiffuseArray				= NULL;
hipArray* gpSpecularArray				= NULL;
hipArray* gpRoughnessArray				= NULL;
hipArray* gpEmissionArray				= NULL;

__constant__ float3		gAaBbMin;
__constant__ float3		gAaBbMax;
__constant__ float3		gInvAaBbMin;
__constant__ float3		gInvAaBbMax;
__constant__ float		gIntensityMin;
__constant__ float		gIntensityMax;
__constant__ float		gIntensityRange;
__constant__ float		gIntensityInvRange;
__constant__ float		gStepSize;
__constant__ float		gStepSizeShadow;
__constant__ float		gDensityScale;
__constant__ float		gGradientDelta;
__constant__ float		gInvGradientDelta;
__constant__ int		gFilmWidth;
__constant__ int		gFilmHeight;
__constant__ int		gFilmNoPixels;
__constant__ int		gFilterWidth;
__constant__ float		gFilterWeights[3];
__constant__ float		gExposure;
__constant__ float		gInvExposure;
__constant__ float		gGamma;
__constant__ float		gInvGamma;
__constant__ float		gDenoiseEnabled;
__constant__ float		gDenoiseWindowRadius;
__constant__ float		gDenoiseInvWindowArea;
__constant__ float		gDenoiseNoise;
__constant__ float		gDenoiseWeightThreshold;
__constant__ float		gDenoiseLerpThreshold;
__constant__ float		gDenoiseLerpC;

#define TF_NO_SAMPLES		256
#define INV_TF_NO_SAMPLES	0.00390625f

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"
#include "Variance.cuh"
#include "NearestIntersection.cuh"

#include "CudaUtilities.h"

void BindDensityBuffer(short* pBuffer, hipExtent Extent)
{
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();

	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexGradientMagnitude));
}

void BindEstimateRgbLdr(CColorRgbaLdr* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<uchar4>();

	gTexEstimateRgbLdr.filterMode = hipFilterModeLinear;     

	HandleCudaError(hipBindTexture2D(0, gTexEstimateRgbLdr, (void*)pBuffer, ChannelDesc, Width, Height, Width * sizeof(uchar4)));
}

void BindTransferFunctionOpacity(CTransferFunction& TransferFunctionOpacity)
{
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	float Opacity[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Opacity[i] = TransferFunctionOpacity.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, TF_NO_SAMPLES * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, ChannelDesc));
}

void UnbindTransferFunctionOpacity(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	gpOpacityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexOpacity));
}

void BindTransferFunctionDiffuse(CTransferFunction& TransferFunctionDiffuse)
{
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	float4 Diffuse[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Diffuse[i].x = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).r;
		Diffuse[i].y = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).g;
		Diffuse[i].z = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, Diffuse, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, ChannelDesc));
}

void UnbindTransferFunctionDiffuse(void)
{
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	gpDiffuseArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
}

void BindTransferFunctionSpecular(CTransferFunction& TransferFunctionSpecular)
{
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	float4 Specular[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Specular[i].x = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).r;
		Specular[i].y = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).g;
		Specular[i].z = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, Specular, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, ChannelDesc));
}

void UnbindTransferFunctionSpecular(void)
{
	HandleCudaError(hipFreeArray(gpSpecularArray));
	gpSpecularArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexSpecular));
}

void BindTransferFunctionRoughness(CTransferFunction& TransferFunctionRoughness)
{
	gTexRoughness.normalized		= true;
	gTexRoughness.filterMode		= hipFilterModeLinear;
	gTexRoughness.addressMode[0]	= hipAddressModeClamp;

	float Roughness[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Roughness[i] = TransferFunctionRoughness.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpRoughnessArray == NULL)
		HandleCudaError(hipMallocArray(&gpRoughnessArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpRoughnessArray, 0, 0, Roughness, TF_NO_SAMPLES * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexRoughness, gpRoughnessArray, ChannelDesc));
}

void UnbindTransferFunctionRoughness(void)
{
	HandleCudaError(hipFreeArray(gpRoughnessArray));
	gpRoughnessArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexRoughness));
}

void BindTransferFunctionEmission(CTransferFunction& TransferFunctionEmission)
{
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	float4 Emission[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Emission[i].x = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).r;
		Emission[i].y = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).g;
		Emission[i].z = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, Emission, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, ChannelDesc));
}

void UnbindTransferFunctionEmission(void)
{
	HandleCudaError(hipFreeArray(gpEmissionArray));
	gpEmissionArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void BindConstants(CScene* pScene)
{
	const float3 AaBbMin = make_float3(pScene->m_BoundingBox.GetMinP().x, pScene->m_BoundingBox.GetMinP().y, pScene->m_BoundingBox.GetMinP().z);
	const float3 AaBbMax = make_float3(pScene->m_BoundingBox.GetMaxP().x, pScene->m_BoundingBox.GetMaxP().y, pScene->m_BoundingBox.GetMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gAaBbMin"), &AaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gAaBbMax"), &AaBbMax, sizeof(float3)));

	const float3 InvAaBbMin = make_float3(pScene->m_BoundingBox.GetInvMinP().x, pScene->m_BoundingBox.GetInvMinP().y, pScene->m_BoundingBox.GetInvMinP().z);
	const float3 InvAaBbMax = make_float3(pScene->m_BoundingBox.GetInvMaxP().x, pScene->m_BoundingBox.GetInvMaxP().y, pScene->m_BoundingBox.GetInvMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gInvAaBbMin"), &InvAaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gInvAaBbMax"), &InvAaBbMax, sizeof(float3)));

	const float IntensityMin		= pScene->m_IntensityRange.GetMin();
	const float IntensityMax		= pScene->m_IntensityRange.GetMax();
	const float IntensityRange		= pScene->m_IntensityRange.GetRange();
	const float IntensityInvRange	= 1.0f / IntensityRange;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gIntensityMin"), &IntensityMin, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gIntensityMax"), &IntensityMax, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gIntensityRange"), &IntensityRange, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gIntensityInvRange"), &IntensityInvRange, sizeof(float)));

	const float StepSize		= pScene->m_StepSizeFactor * pScene->m_GradientDelta;
	const float StepSizeShadow	= pScene->m_StepSizeFactorShadow * pScene->m_GradientDelta;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gStepSize"), &StepSize, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gStepSizeShadow"), &StepSizeShadow, sizeof(float)));

	const float DensityScale = pScene->m_DensityScale;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDensityScale"), &DensityScale, sizeof(float)));
	
	const float GradientDelta		= pScene->m_GradientDelta;
	const float InvGradientDelta	= 1.0f / GradientDelta;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gGradientDelta"), &GradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gInvGradientDelta"), &InvGradientDelta, sizeof(float)));
	
	const int FilmWidth		= pScene->m_Camera.m_Film.GetWidth();
	const int Filmheight	= pScene->m_Camera.m_Film.GetHeight();
	const int FilmNoPixels	= pScene->m_Camera.m_Film.m_Resolution.GetNoElements();

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gFilmWidth"), &FilmWidth, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gFilmHeight"), &Filmheight, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gFilmNoPixels"), &FilmNoPixels, sizeof(int)));

	const int FilterWidth = 2;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gFilterWidth"), &FilterWidth, sizeof(int)));

	const float FilterWeights[3] = { 0.11411459588254977f, 0.08176668094332218f, 0.03008028089187349f };

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gFilterWeights"), &FilterWeights, 3 * sizeof(float)));

	const float Gamma		= pScene->m_Camera.m_Film.m_Gamma;
	const float InvGamma	= 1.0f / Gamma;
	const float Exposure	= pScene->m_Camera.m_Film.m_Exposure;
	const float InvExposure	= 1.0f / Exposure;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gExposure"), &Exposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gInvExposure"), &InvExposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gGamma"), &Gamma, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gInvGamma"), &InvGamma, sizeof(float)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoiseEnabled"), &pScene->m_DenoiseParams.m_Enabled, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoiseWindowRadius"), &pScene->m_DenoiseParams.m_WindowRadius, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoiseInvWindowArea"), &pScene->m_DenoiseParams.m_InvWindowArea, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoiseNoise"), &pScene->m_DenoiseParams.m_Noise, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoiseWeightThreshold"), &pScene->m_DenoiseParams.m_WeightThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoiseLerpThreshold"), &pScene->m_DenoiseParams.m_LerpThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoiseLerpC"), &pScene->m_DenoiseParams.m_LerpC, sizeof(float)));
}

void Render(const int& Type, CScene& Scene, CCudaFrameBuffers& CudaFrameBuffers, int N, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CScene* pDevScene = NULL;

	HandleCudaError(hipMalloc(&pDevScene, sizeof(CScene)));
	HandleCudaError(hipMemcpy(pDevScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));

	if (Scene.m_Camera.m_Focus.m_Type == 0)
		Scene.m_Camera.m_Focus.m_FocalDistance = NearestIntersection(pDevScene);

	HandleCudaError(hipMemcpy(pDevScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));

	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(&Scene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}

		case 1:
		{
			MultipleScattering(&Scene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());
	
 	CCudaTimer TmrBlur;
	BlurImageXyz(&Scene, pDevScene, CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevEstFrameBlurXyz);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	Estimate(&Scene, pDevScene, CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevAccEstXyz, CudaFrameBuffers.m_pDevEstXyz, CudaFrameBuffers.m_pDevEstRgbaLdr, N);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	CCudaTimer TmrDenoise;
	Denoise(&Scene, pDevScene, CudaFrameBuffers.m_pDevEstRgbaLdr, CudaFrameBuffers.m_pDevRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());

	HandleCudaError(hipFree(pDevScene));
}