#include "hip/hip_runtime.h"

#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, 3, hipReadModeElementType>			gTexExtinction;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexEstimateRgbLdr;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexOpacity;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexDiffuse;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexSpecular;

hipArray* gpDensityArray			= NULL;
hipArray* gpGradientMagnitudeArray	= NULL;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"
#include "Variance.cuh"

#include "CudaUtilities.h"

void BindDensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc);
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc);
}

void UnbindDensityBuffer(void)
{
	hipFreeArray(gpDensityArray);
	gpDensityArray = NULL;
	hipUnbindTexture(gTexDensity);
}

void UnbindGradientMagnitudeBuffer(void)
{
	hipFreeArray(gpGradientMagnitudeArray);
	gpGradientMagnitudeArray = NULL;
	hipUnbindTexture(gTexGradientMagnitude);
}

void BindEstimateRgbLdr(CColorRgbaLdr* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<uchar4>();

	hipBindTexture2D(0, gTexEstimateRgbLdr, (void*)pBuffer, ChannelDesc, Width, Height, Width * sizeof(uchar4));
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, CCudaFrameBuffers& CudaFrameBuffers, int N, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());
	
 	CCudaTimer TmrBlur;
	BlurImageXyz(CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 5.0f);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	Estimate(pScene, pDevScene, CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevAccEstXyz, CudaFrameBuffers.m_pDevEstXyz, CudaFrameBuffers.m_pDevEstRgbaLdr, N);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());
	/**/

	CCudaTimer TmrDenoise;
	Denoise(pScene, pDevScene, CudaFrameBuffers.m_pDevEstRgbaLdr, CudaFrameBuffers.m_pDevRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
	
}