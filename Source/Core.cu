#include "hip/hip_runtime.h"

#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, 3, hipReadModeElementType>			gTexExtinction;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexEstimateRgbLdr;
texture<float4, 1, hipReadModeElementType>			gTexOpacity;
texture<float4, 1, hipReadModeElementType>			gTexDiffuse;
texture<float4, 1, hipReadModeElementType>			gTexSpecular;

hipArray* gpDensityArray			= NULL;
hipArray* gpGradientMagnitudeArray	= NULL;

hipArray* gpOpacityArray = NULL;

#define TF_NO_SAMPLES		256
#define INV_TF_NO_SAMPLES	1.0f / (float)TF_NO_SAMPLES

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"
#include "Variance.cuh"

#include "CudaUtilities.h"

void BindDensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc);
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc);
}

void UnbindDensityBuffer(void)
{
	hipFreeArray(gpDensityArray);
	gpDensityArray = NULL;
	hipUnbindTexture(gTexDensity);
}

void UnbindGradientMagnitudeBuffer(void)
{
	hipFreeArray(gpGradientMagnitudeArray);
	gpGradientMagnitudeArray = NULL;
	hipUnbindTexture(gTexGradientMagnitude);
}

void BindEstimateRgbLdr(CColorRgbaLdr* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<uchar4>();

	hipBindTexture2D(0, gTexEstimateRgbLdr, (void*)pBuffer, ChannelDesc, Width, Height, Width * sizeof(uchar4));
}

void BindOpacity(CTransferFunction& Opacity)
{
	float4 Val[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Val[i].x = Opacity.F(i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	
	if (gpOpacityArray == NULL)
		hipMallocArray(&gpOpacityArray, &channelDesc, TF_NO_SAMPLES, 1);

	hipMemcpyToArray(gpOpacityArray, 0, 0, Val, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice);

	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0] = hipAddressModeClamp;
	
		hipBindTextureToArray(gTexOpacity, gpOpacityArray, channelDesc);
		
/*
	if (gpOpacity == NULL)
		hipMalloc(&gpOpacity, TF_NO_SAMPLES * sizeof(float));

	hipMemcpy(gpOpacity, &Val, TF_NO_SAMPLES * sizeof(float), hipMemcpyHostToDevice);


	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeWrap;

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	
	hipBindTexture(0, gTexOpacity, gpOpacity, ChannelDesc, TF_NO_SAMPLES * sizeof(float));*/  
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, CCudaFrameBuffers& CudaFrameBuffers, int N, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());
	
 	CCudaTimer TmrBlur;
	BlurImageXyz(CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 5.0f);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	Estimate(pScene, pDevScene, CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevAccEstXyz, CudaFrameBuffers.m_pDevEstXyz, CudaFrameBuffers.m_pDevEstRgbaLdr, N);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	CCudaTimer TmrDenoise;
	Denoise(pScene, pDevScene, CudaFrameBuffers.m_pDevEstRgbaLdr, CudaFrameBuffers.m_pDevRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
}