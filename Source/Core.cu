#include "hip/hip_runtime.h"

#include "Core.cuh"

texture<float, 3, hipReadModeElementType>			gTexDensity;
texture<float, 3, hipReadModeElementType>			gTexExtinction;
texture<float, 3, hipReadModeElementType>			gTexGradientMagnitude;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexEstimateRgbLdr;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "ComputeEstimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

#include "CudaUtilities.h"

void BindDensityBuffer(float* pBuffer, hipExtent Extent)
{
	hipArray* pArray = NULL;

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&pArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr				= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(float), Extent.width, Extent.height);
	CopyParams.dstArray				= pArray;
	CopyParams.extent				= Extent;
	CopyParams.kind					= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	// Set texture parameters
	gTexDensity.normalized			= true;
	gTexDensity.filterMode			= hipFilterModeLinear;      
	gTexDensity.addressMode[0]		= hipAddressModeClamp;  
	gTexDensity.addressMode[1]		= hipAddressModeClamp;
//  	gTexDensity.addressMode[2]		= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, pArray, ChannelDesc);
}

void BindExtinctionBuffer(float* pBuffer, hipExtent Extent)
{
	hipArray* pArray;

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&pArray, &ChannelDesc, Extent);
	
	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr				= make_hipPitchedPtr((void*)pBuffer, Extent.width * sizeof(float), Extent.width, Extent.height);
	CopyParams.dstArray				= pArray;
	CopyParams.extent				= Extent;
	CopyParams.kind					= hipMemcpyHostToDevice;

	hipMemcpy3D(&CopyParams);

	gTexExtinction.normalized		= true;
	gTexExtinction.filterMode		= hipFilterModePoint;
	gTexExtinction.addressMode[0]	= hipAddressModeClamp;
	gTexExtinction.addressMode[1]	= hipAddressModeClamp;
// 	gTexExtinction.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexExtinction, pArray, ChannelDesc);
}

void BindGradientMagnitudeBuffer(float* pBuffer, hipExtent Extent)
{
	hipArray* pArray;

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&pArray, &ChannelDesc, Extent);
	
	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr((void*)pBuffer, Extent.width * sizeof(float), Extent.width, Extent.height);
	CopyParams.dstArray = pArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;

	CUDA_SAFE_CALL(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModePoint;
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
// 	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, pArray, ChannelDesc);
}

void BindEstimateRgbLdr(unsigned char* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<uchar4>();

	hipBindTexture2D(0, gTexEstimateRgbLdr, pBuffer, ChannelDesc, Width, Height, Width * sizeof(uchar4));
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, CColorRgbaLdr* pDevEstRgbaLdr, unsigned char* pDevEstRgbLdrDisp, int N, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
			{
				SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
				HandleCudaError(hipGetLastError());
				break;
			}

		case 1:
			{
				MultipleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
				HandleCudaError(hipGetLastError());
				break;
			}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());

 	CCudaTimer TmrBlur;
	BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 1.3f);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	ComputeEstimate(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY(), pDevEstFrameXyz, pDevAccEstXyz, N, pScene->m_Camera.m_Film.m_Exposure, pDevEstRgbaLdr);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	CCudaTimer TmrDenoise;
	Denoise(pScene, pDevScene, pDevEstRgbaLdr, (CColorRgbLdr*)pDevEstRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
}