#include "hip/hip_runtime.h"

#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, 3, hipReadModeElementType>			gTexExtinction;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexEstimateRgbLdr;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexOpacity;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexDiffuse;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexSpecular;

hipArray* gpDensityArray			= NULL;
hipArray* gpGradientMagnitudeArray	= NULL;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"
#include "Variance.cuh"

#include "CudaUtilities.h"

void BindDensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc);
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc);
}

void UnbindDensityBuffer(void)
{
	hipFreeArray(gpDensityArray);
	gpDensityArray = NULL;
	hipUnbindTexture(gTexDensity);
}

void UnbindGradientMagnitudeBuffer(void)
{
	hipFreeArray(gpGradientMagnitudeArray);
	gpGradientMagnitudeArray = NULL;
	hipUnbindTexture(gTexGradientMagnitude);
}

void BindEstimateRgbLdr(CColorRgbaLdr* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<uchar4>();

	hipBindTexture2D(0, gTexEstimateRgbLdr, (void*)pBuffer, ChannelDesc, Width, Height, Width * sizeof(uchar4));
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, CColorXyz* pDevEstXyz, CColorRgbaLdr* pDevEstRgbaLdr, CColorRgbLdr* pDevEstRgbLdrDisp, int N, CVariance* pDevVariance, float* pVariance, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());

 	CCudaTimer TmrBlur;
	BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 3.0f);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	Estimate(pScene, pDevScene, pDevEstFrameXyz, pDevAccEstXyz, pDevEstXyz, pDevEstRgbaLdr, N, pDevVariance);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	CCudaTimer TmrDenoise;
	Denoise(pScene, pDevScene, pDevEstRgbaLdr, pDevEstRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
}