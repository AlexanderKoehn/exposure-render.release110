
#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat >	gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat >	gTexExtinction;
texture<short, 3, hipReadModeNormalizedFloat >	gTexGradientMagnitude;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "ComputeEstimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

void BindDensityVolume(short* densityBuffer, hipExtent densityBufferSize)
{
	hipArray* gpDensity = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensity, &ChannelDesc, densityBufferSize);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(densityBuffer, densityBufferSize.width * sizeof(short), densityBufferSize.width, densityBufferSize.height);
	copyParams.dstArray				= gpDensity;
	copyParams.extent				= densityBufferSize;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexDensity.normalized			= true;
	gTexDensity.filterMode			= hipFilterModeLinear;      
	gTexDensity.addressMode[0]		= hipAddressModeClamp;  
	gTexDensity.addressMode[1]		= hipAddressModeClamp;
 	gTexDensity.addressMode[2]		= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, gpDensity, ChannelDesc);
}

void BindExtinctionVolume(float* extinction, hipExtent extinctionSize)
{
	hipArray* volArray;
	hipExtent volExtent = make_hipExtent(extinctionSize.width, extinctionSize.height, extinctionSize.depth);
	hipChannelFormatDesc volChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&volArray, &volChannelDesc, volExtent);
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr = make_hipPitchedPtr((void*)extinction, volExtent.width * sizeof(float), volExtent.width, volExtent.height);
	copyParams.dstArray = volArray;
	copyParams.extent = volExtent;
	copyParams.kind = hipMemcpyHostToDevice;
	CUDA_SAFE_CALL( hipMemcpy3D(&copyParams));

	gTexExtinction.normalized = true;
	gTexExtinction.filterMode = hipFilterModePoint;
	gTexExtinction.addressMode[0] = hipAddressModeClamp;
	gTexExtinction.addressMode[1] = hipAddressModeClamp;

	hipBindTextureToArray( gTexExtinction, volArray, volChannelDesc);
}

void BindGradientMagnitudeVolume(short* pBuffer, hipExtent VolumeSize)
{
	hipArray* pVolumeArray;

	hipChannelFormatDesc VolumeChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&pVolumeArray, &VolumeChannelDesc, VolumeSize);
	
	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr((void*)pBuffer, VolumeSize.width * sizeof(short), VolumeSize.width, VolumeSize.height);
	CopyParams.dstArray = pVolumeArray;
	CopyParams.extent	= VolumeSize;
	CopyParams.kind		= hipMemcpyHostToDevice;

	CUDA_SAFE_CALL(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModePoint;
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, pVolumeArray, VolumeChannelDesc);
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, unsigned char* pDevEstRgbLdr, unsigned char* pDevEstRgbLdrDisp, int N)
{
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			break;
		}
	}

	BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 1.3f);
  	ComputeEstimate(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY(), pDevEstFrameXyz, pDevAccEstXyz, N, pScene->m_Camera.m_Film.m_Exposure, pDevEstRgbLdr);
	Denoise(pScene, pDevScene, (CColorRgbLdr*)pDevEstRgbLdr, (CColorRgbLdr*)pDevEstRgbLdrDisp);
}