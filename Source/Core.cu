
#include "Core.cuh"

texture<float, 3, hipReadModeElementType>	gTexDensity;
texture<float, 3, hipReadModeElementType>	gTexExtinction;

#include "Blur.cuh"
#include "ComputeEstimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

void BindDensityVolume(float* densityBuffer, hipExtent densityBufferSize)
{
	/*hipArray* gpDensity = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&gpDensity, &ChannelDesc, Size);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(pDensityBuffer, Size.width * sizeof(float), Size.width, Size.height);
	copyParams.dstArray				= gpDensity;
	copyParams.extent				= Size;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
 	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, gpDensity, ChannelDesc);
	*/

	 hipArray* densityArray;
    hipChannelFormatDesc densityBufferChannelDesc = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL(hipMalloc3DArray(&densityArray, &densityBufferChannelDesc, densityBufferSize));
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)densityBuffer, densityBufferSize.width * sizeof(float), densityBufferSize.width, densityBufferSize.height);
    copyParams.dstArray = densityArray;
    copyParams.extent = densityBufferSize;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));

    gTexDensity.normalized = true;
    gTexDensity.filterMode = hipFilterModeLinear;
    gTexDensity.addressMode[0] = hipAddressModeClamp;
    gTexDensity.addressMode[1] = hipAddressModeClamp;

    hipBindTextureToArray( gTexDensity, densityArray, densityBufferChannelDesc);
}

void BindExtinctionVolume(float* extinction, hipExtent extinctionSize)
{
/*
	hipArray* gpExtinction = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&gpExtinction, &ChannelDesc, Size);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(pExtinctionBuffer, Size.width * sizeof(float), Size.width, Size.height);
	copyParams.dstArray				= gpExtinction;
	copyParams.extent				= Size;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexExtinction.normalized		= true;
	gTexExtinction.filterMode		= hipFilterModePoint;      
	gTexExtinction.addressMode[0]	= hipAddressModeClamp;  
	gTexExtinction.addressMode[1]	= hipAddressModeClamp;
// 	gTexExtinction.addressMode[2]	= hipAddressModeClamp;*/

	// Bind array to 3D texture
//	hipBindTextureToArray(gTexExtinction, gpExtinction, ChannelDesc);
	hipArray* volArray;
	 hipExtent volExtent = make_hipExtent(extinctionSize.width, extinctionSize.height, extinctionSize.depth);
  hipChannelFormatDesc volChannelDesc = hipCreateChannelDesc<float>();
  hipMalloc3DArray(&volArray, &volChannelDesc, volExtent);
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr((void*)extinction, volExtent.width * sizeof(float), volExtent.width, volExtent.height);
  copyParams.dstArray = volArray;
  copyParams.extent = volExtent;
  copyParams.kind = hipMemcpyHostToDevice;
  CUDA_SAFE_CALL( hipMemcpy3D(&copyParams));

  gTexExtinction.normalized = true;
  gTexExtinction.filterMode = hipFilterModePoint;
  gTexExtinction.addressMode[0] = hipAddressModeClamp;
  gTexExtinction.addressMode[1] = hipAddressModeClamp;

  hipBindTextureToArray( gTexExtinction, volArray, volChannelDesc);
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, unsigned char* pDevEstRgbLdr, int N)
{
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 1.3f);
  			ComputeEstimate(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY(), pDevEstFrameXyz, pDevAccEstXyz, N, 500.0f, pDevEstRgbLdr);
			
			break;
		}

		case 1:
		{
			break;
		}
	}
}