
#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat>	gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>	gTexExtinction;

#include "Blur.cuh"
#include "ComputeEstimate.cuh"
#include "Random.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

void BindDensityVolume(short* pDensityBuffer, hipExtent Size)
{
	hipArray* gpDensity = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensity, &ChannelDesc, Size);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(pDensityBuffer, Size.width * sizeof(short), Size.width, Size.height);
	copyParams.dstArray				= gpDensity;
	copyParams.extent				= Size;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
 	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, gpDensity, ChannelDesc);
}

void BindExtinctionVolume(short* pExtinctionBuffer, hipExtent Size)
{
	hipArray* gpExtinction = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpExtinction, &ChannelDesc, Size);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(pExtinctionBuffer, Size.width * sizeof(short), Size.width, Size.height);
	copyParams.dstArray				= gpExtinction;
	copyParams.extent				= Size;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexExtinction.normalized		= true;
	gTexExtinction.filterMode		= hipFilterModePoint;      
	gTexExtinction.addressMode[0]	= hipAddressModeClamp;  
	gTexExtinction.addressMode[1]	= hipAddressModeClamp;
// 	gTexExtinction.addressMode[2]	= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexExtinction, gpExtinction, ChannelDesc);
}