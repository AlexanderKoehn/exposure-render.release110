#include "hip/hip_runtime.h"

#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, 3, hipReadModeElementType>			gTexExtinction;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexEstimateRgbLdr;
texture<float, 1, hipReadModeElementType>			gTexOpacity;
texture<float4, 1, hipReadModeElementType>			gTexDiffuse;
texture<float4, 1, hipReadModeElementType>			gTexSpecular;
texture<float, 1, hipReadModeElementType>			gTexRoughness;
texture<float4, 1, hipReadModeElementType>			gTexEmission;

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
hipArray* gpOpacityArray				= NULL;
hipArray* gpDiffuseArray				= NULL;
hipArray* gpSpecularArray				= NULL;
hipArray* gpRoughnessArray				= NULL;
hipArray* gpEmissionArray				= NULL;

#define TF_NO_SAMPLES		256
#define INV_TF_NO_SAMPLES	0.00390625f

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"
#include "Variance.cuh"

#include "CudaUtilities.h"

void BindDensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc);
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc);
}

void UnbindDensityBuffer(void)
{
	hipFreeArray(gpDensityArray);
	gpDensityArray = NULL;
	hipUnbindTexture(gTexDensity);
}

void UnbindGradientMagnitudeBuffer(void)
{
	hipFreeArray(gpGradientMagnitudeArray);
	gpGradientMagnitudeArray = NULL;
	hipUnbindTexture(gTexGradientMagnitude);
}

void BindEstimateRgbLdr(CColorRgbaLdr* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<uchar4>();

	hipBindTexture2D(0, gTexEstimateRgbLdr, (void*)pBuffer, ChannelDesc, Width, Height, Width * sizeof(uchar4));
}

void BindTransferFunctions(CTransferFunctions& TransferFunctions)
{
	hipChannelFormatDesc ChannelDesc;

	gTexOpacity.normalized			= true;
	gTexDiffuse.normalized			= true;
	gTexSpecular.normalized			= true;
	gTexRoughness.normalized		= true;
	gTexEmission.normalized			= true;

	gTexOpacity.filterMode			= hipFilterModeLinear;
	gTexDiffuse.filterMode			= hipFilterModeLinear;
	gTexSpecular.filterMode			= hipFilterModeLinear;
	gTexRoughness.filterMode		= hipFilterModeLinear;
	gTexEmission.filterMode			= hipFilterModeLinear;

	gTexOpacity.addressMode[0]		= hipAddressModeClamp;
	gTexDiffuse.addressMode[0]		= hipAddressModeClamp;
	gTexSpecular.addressMode[0]		= hipAddressModeClamp;
	gTexRoughness.addressMode[0]	= hipAddressModeClamp;
	gTexEmission.addressMode[0]		= hipAddressModeClamp;

	// Opacity
	float Opacity[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Opacity[i] = TransferFunctions.m_Opacity.F((float)i * INV_TF_NO_SAMPLES).r;
	
	ChannelDesc = hipCreateChannelDesc<float>();

	if (gpOpacityArray == NULL)
		hipMallocArray(&gpOpacityArray, &ChannelDesc, TF_NO_SAMPLES, 1);

	hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, TF_NO_SAMPLES * sizeof(float),  hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexOpacity, gpOpacityArray, ChannelDesc);
//	float4* pDevOpacity = NULL;
//	hipMalloc(&pDevOpacity, 256 * sizeof(float4));
//	hipBindTexture(0, gTexOpacity, pDevOpacity, ChannelDesc, 256 * sizeof(float4));

	// Diffuse
	float4 Diffuse[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Diffuse[i].x = TransferFunctions.m_Diffuse.F((float)i * INV_TF_NO_SAMPLES).r;
		Diffuse[i].y = TransferFunctions.m_Diffuse.F((float)i * INV_TF_NO_SAMPLES).g;
		Diffuse[i].z = TransferFunctions.m_Diffuse.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpDiffuseArray == NULL)
		hipMallocArray(&gpDiffuseArray, &ChannelDesc, TF_NO_SAMPLES, 1);

	hipMemcpyToArray(gpDiffuseArray, 0, 0, Diffuse, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, ChannelDesc);

	// Specular
	float4 Specular[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Specular[i].x = TransferFunctions.m_Specular.F((float)i * INV_TF_NO_SAMPLES).r;
		Specular[i].y = TransferFunctions.m_Specular.F((float)i * INV_TF_NO_SAMPLES).g;
		Specular[i].z = TransferFunctions.m_Specular.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpSpecularArray == NULL)
		hipMallocArray(&gpSpecularArray, &ChannelDesc, TF_NO_SAMPLES, 1);

	hipMemcpyToArray(gpSpecularArray, 0, 0, Specular, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexSpecular, gpSpecularArray, ChannelDesc);

	// Roughness
	float Roughness[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Roughness[i] = TransferFunctions.m_Roughness.F((float)i * INV_TF_NO_SAMPLES).r;
	
	if (gpRoughnessArray == NULL)
		hipMallocArray(&gpRoughnessArray, &ChannelDesc, TF_NO_SAMPLES, 1);

	hipMemcpyToArray(gpRoughnessArray, 0, 0, Roughness, TF_NO_SAMPLES * sizeof(float),  hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexRoughness, gpRoughnessArray, ChannelDesc);

	// Emission
	float4 Emission[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Emission[i].x = TransferFunctions.m_Emission.F((float)i * INV_TF_NO_SAMPLES).r;
		Emission[i].y = TransferFunctions.m_Emission.F((float)i * INV_TF_NO_SAMPLES).g;
		Emission[i].z = TransferFunctions.m_Emission.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpEmissionArray == NULL)
		hipMallocArray(&gpEmissionArray, &ChannelDesc, TF_NO_SAMPLES, 1);

	hipMemcpyToArray(gpEmissionArray, 0, 0, Emission, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexEmission, gpEmissionArray, ChannelDesc);
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, CCudaFrameBuffers& CudaFrameBuffers, int N, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, CudaFrameBuffers.m_pDevSeeds, CudaFrameBuffers.m_pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());
	
 	CCudaTimer TmrBlur;
	BlurImageXyz(CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 5.0f);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	Estimate(pScene, pDevScene, CudaFrameBuffers.m_pDevEstFrameXyz, CudaFrameBuffers.m_pDevAccEstXyz, CudaFrameBuffers.m_pDevEstXyz, CudaFrameBuffers.m_pDevEstRgbaLdr, N);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	CCudaTimer TmrDenoise;
	Denoise(pScene, pDevScene, CudaFrameBuffers.m_pDevEstRgbaLdr, CudaFrameBuffers.m_pDevRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
}