
#include "Core.cuh"

texture<float, 3, hipReadModeElementType>				gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>			gTexExtinction;
texture<float, 3, hipReadModeElementType>				gTexGradientMagnitude;
texture<unsigned char, 3, hipReadModeNormalizedFloat>	gTexEstimateRgbLdr;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "ComputeEstimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

#include "CudaUtilities.h"

void BindDensityVolume(float* densityBuffer, hipExtent Extent)
{
	hipArray* pArray = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&pArray, &ChannelDesc, Extent);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(densityBuffer, Extent.width * sizeof(float), Extent.width, Extent.height);
	copyParams.dstArray				= pArray;
	copyParams.extent				= Extent;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexDensity.normalized			= true;
	gTexDensity.filterMode			= hipFilterModeLinear;      
	gTexDensity.addressMode[0]		= hipAddressModeClamp;  
	gTexDensity.addressMode[1]		= hipAddressModeClamp;
 	gTexDensity.addressMode[2]		= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, pArray, ChannelDesc);
}

void BindExtinctionVolume(float* extinction, hipExtent Extent)
{
	hipArray* volArray;
	hipExtent volExtent = make_hipExtent(Extent.width, Extent.height, Extent.depth);
	hipChannelFormatDesc volChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&volArray, &volChannelDesc, volExtent);
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr = make_hipPitchedPtr((void*)extinction, Extent.width * sizeof(float), Extent.width, Extent.height);
	copyParams.dstArray = volArray;
	copyParams.extent = Extent;
	copyParams.kind = hipMemcpyHostToDevice;
	CUDA_SAFE_CALL( hipMemcpy3D(&copyParams));

	gTexExtinction.normalized = true;
	gTexExtinction.filterMode = hipFilterModePoint;
	gTexExtinction.addressMode[0] = hipAddressModeClamp;
	gTexExtinction.addressMode[1] = hipAddressModeClamp;

	hipBindTextureToArray(gTexExtinction, volArray, volChannelDesc);
}

void BindGradientMagnitudeVolume(float* pBuffer, hipExtent Extent)
{
	hipArray* pVolumeArray;

	hipChannelFormatDesc VolumeChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&pVolumeArray, &VolumeChannelDesc, Extent);
	
	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr((void*)pBuffer, Extent.width * sizeof(float), Extent.width, Extent.height);
	CopyParams.dstArray = pVolumeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;

	CUDA_SAFE_CALL(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModePoint;
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, pVolumeArray, VolumeChannelDesc);
}

void BindEstimateRgbLdr(unsigned char* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<unsigned char>();

	hipBindTexture2D(0, gTexEstimateRgbLdr, pBuffer, ChannelDesc, Width, Height, Width * sizeof(unsigned char));
}


void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, unsigned char* pDevEstRgbLdr, unsigned char* pDevEstRgbLdrDisp, int N, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());

 	CCudaTimer TmrBlur;
	BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 1.3f);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	ComputeEstimate(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY(), pDevEstFrameXyz, pDevAccEstXyz, N, pScene->m_Camera.m_Film.m_Exposure, pDevEstRgbLdr);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	CCudaTimer TmrDenoise;
//	Denoise(pScene, pDevScene, (CColorRgbLdr*)pDevEstRgbLdr, (CColorRgbLdr*)pDevEstRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
}