
#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat >	gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat >	gTexExtinction;

#include "Blur.cuh"
#include "ComputeEstimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

void BindDensityVolume(short* densityBuffer, hipExtent densityBufferSize)
{
	hipArray* gpDensity = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensity, &ChannelDesc, densityBufferSize);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(densityBuffer, densityBufferSize.width * sizeof(short), densityBufferSize.width, densityBufferSize.height);
	copyParams.dstArray				= gpDensity;
	copyParams.extent				= densityBufferSize;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexDensity.normalized			= true;
	gTexDensity.filterMode			= hipFilterModeLinear;      
	gTexDensity.addressMode[0]		= hipAddressModeClamp;  
	gTexDensity.addressMode[1]		= hipAddressModeClamp;
 	gTexDensity.addressMode[2]		= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, gpDensity, ChannelDesc);
}

void BindExtinctionVolume(float* extinction, hipExtent extinctionSize)
{
/*
	hipArray* gpExtinction = NULL;

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&gpExtinction, &ChannelDesc, Size);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(pExtinctionBuffer, Size.width * sizeof(float), Size.width, Size.height);
	copyParams.dstArray				= gpExtinction;
	copyParams.extent				= Size;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexExtinction.normalized		= true;
	gTexExtinction.filterMode		= hipFilterModePoint;      
	gTexExtinction.addressMode[0]	= hipAddressModeClamp;  
	gTexExtinction.addressMode[1]	= hipAddressModeClamp;
// 	gTexExtinction.addressMode[2]	= hipAddressModeClamp;*/

	// Bind array to 3D texture
//	hipBindTextureToArray(gTexExtinction, gpExtinction, ChannelDesc);
	hipArray* volArray;
	 hipExtent volExtent = make_hipExtent(extinctionSize.width, extinctionSize.height, extinctionSize.depth);
  hipChannelFormatDesc volChannelDesc = hipCreateChannelDesc<float>();
  hipMalloc3DArray(&volArray, &volChannelDesc, volExtent);
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr((void*)extinction, volExtent.width * sizeof(float), volExtent.width, volExtent.height);
  copyParams.dstArray = volArray;
  copyParams.extent = volExtent;
  copyParams.kind = hipMemcpyHostToDevice;
  CUDA_SAFE_CALL( hipMemcpy3D(&copyParams));

  gTexExtinction.normalized = true;
  gTexExtinction.filterMode = hipFilterModePoint;
  gTexExtinction.addressMode[0] = hipAddressModeClamp;
  gTexExtinction.addressMode[1] = hipAddressModeClamp;

  hipBindTextureToArray( gTexExtinction, volArray, volChannelDesc);
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, unsigned char* pDevEstRgbLdr, int N)
{
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 1.3f);
  			ComputeEstimate(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY(), pDevEstFrameXyz, pDevAccEstXyz, N, 500.0f, pDevEstRgbLdr);
			
			break;
		}

		case 1:
		{
			break;
		}
	}
}