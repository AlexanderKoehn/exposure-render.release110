
#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat >	gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat >	gTexExtinction;
texture<short, 3, hipReadModeNormalizedFloat >	gTexGradientMagnitude;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "ComputeEstimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

#include "CudaUtilities.h"

void BindDensityVolume(short* densityBuffer, hipExtent densityBufferSize)
{
	hipArray* pArray = NULL;

	hipExtent volExtent = make_hipExtent(densityBufferSize.width, densityBufferSize.height, densityBufferSize.depth);

	// create 3D array
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&pArray, &ChannelDesc, densityBufferSize);

	// copy data to 3D array
	hipMemcpy3DParms copyParams	= {0};
	copyParams.srcPtr				= make_hipPitchedPtr(densityBuffer, volExtent.width * sizeof(short), volExtent.width, volExtent.height);
	copyParams.dstArray				= pArray;
	copyParams.extent				= volExtent;
	copyParams.kind					= hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// Set texture parameters
	gTexDensity.normalized			= true;
	gTexDensity.filterMode			= hipFilterModeLinear;      
	gTexDensity.addressMode[0]		= hipAddressModeClamp;  
	gTexDensity.addressMode[1]		= hipAddressModeClamp;
 	gTexDensity.addressMode[2]		= hipAddressModeClamp;

	// Bind array to 3D texture
	hipBindTextureToArray(gTexDensity, pArray, ChannelDesc);
}

void BindExtinctionVolume(float* extinction, hipExtent extinctionSize)
{
	hipArray* volArray;
	hipExtent volExtent = make_hipExtent(extinctionSize.width, extinctionSize.height, extinctionSize.depth);
	hipChannelFormatDesc volChannelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&volArray, &volChannelDesc, volExtent);
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr = make_hipPitchedPtr((void*)extinction, volExtent.width * sizeof(float), volExtent.width, volExtent.height);
	copyParams.dstArray = volArray;
	copyParams.extent = volExtent;
	copyParams.kind = hipMemcpyHostToDevice;
	CUDA_SAFE_CALL( hipMemcpy3D(&copyParams));

	gTexExtinction.normalized = true;
	gTexExtinction.filterMode = hipFilterModePoint;
	gTexExtinction.addressMode[0] = hipAddressModeClamp;
	gTexExtinction.addressMode[1] = hipAddressModeClamp;

	hipBindTextureToArray( gTexExtinction, volArray, volChannelDesc);
}

void BindGradientMagnitudeVolume(short* pBuffer, hipExtent VolumeSize)
{
	hipArray* pVolumeArray;

	hipExtent volExtent = make_hipExtent(VolumeSize.width, VolumeSize.height, VolumeSize.depth);

	hipChannelFormatDesc VolumeChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&pVolumeArray, &VolumeChannelDesc, VolumeSize);
	
	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr((void*)pBuffer, volExtent.width * sizeof(short), volExtent.width, volExtent.height);
	CopyParams.dstArray = pVolumeArray;
	CopyParams.extent	= volExtent;
	CopyParams.kind		= hipMemcpyHostToDevice;

	CUDA_SAFE_CALL(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModePoint;
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, pVolumeArray, VolumeChannelDesc);
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, unsigned char* pDevEstRgbLdr, unsigned char* pDevEstRgbLdrDisp, int N, CTiming& Casting, CTiming& Blur, CTiming& ToneMap)
{
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			break;
		}
	}

// 	CCudaTimer TmrBlur;
	BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 1.3f);
// 	Blur.AddDuration(TmrBlur.StopTimer());

  	ComputeEstimate(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY(), pDevEstFrameXyz, pDevAccEstXyz, N, pScene->m_Camera.m_Film.m_Exposure, pDevEstRgbLdr);
	Denoise(pScene, pDevScene, (CColorRgbLdr*)pDevEstRgbLdr, (CColorRgbLdr*)pDevEstRgbLdrDisp);
}