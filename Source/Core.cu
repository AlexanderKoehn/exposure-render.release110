#include "hip/hip_runtime.h"

#include "Core.cuh"

texture<short, 3, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, 3, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, 3, hipReadModeElementType>			gTexExtinction;
texture<uchar4, 2, hipReadModeNormalizedFloat>		gTexEstimateRgbLdr;

hipArray* gpDensityArray			= NULL;
hipArray* gpGradientMagnitudeArray	= NULL;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "ComputeEstimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "MultipleScattering.cuh"

#include "CudaUtilities.h"

void BindDensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc);
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc);
}

void UnbindDensityBuffer(void)
{
	hipFreeArray(gpDensityArray);
	gpDensityArray = NULL;
	hipUnbindTexture(gTexDensity);
}

void UnbindGradientMagnitudeBuffer(void)
{
	hipFreeArray(gpGradientMagnitudeArray);
	gpGradientMagnitudeArray = NULL;
	hipUnbindTexture(gTexGradientMagnitude);
}

void BindEstimateRgbLdr(unsigned char* pBuffer, int Width, int Height)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<uchar4>();

	hipBindTexture2D(0, gTexEstimateRgbLdr, pBuffer, ChannelDesc, Width, Height, Width * sizeof(uchar4));
}

void Render(const int& Type, CScene* pScene, CScene* pDevScene, unsigned int* pSeeds, CColorXyz* pDevEstFrameXyz, CColorXyz* pDevEstFrameBlurXyz, CColorXyz* pDevAccEstXyz, CColorRgbaLdr* pDevEstRgbaLdr, unsigned char* pDevEstRgbLdrDisp, int N, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}

		case 1:
		{
			MultipleScattering(pScene, pDevScene, pSeeds, pDevEstFrameXyz);
			HandleCudaError(hipGetLastError());
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());

 	CCudaTimer TmrBlur;
	BlurImageXyz(pDevEstFrameXyz, pDevEstFrameBlurXyz, CResolution2D(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY()), 3.5f);
	HandleCudaError(hipGetLastError());
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	ComputeEstimate(pScene->m_Camera.m_Film.m_Resolution.GetResX(), pScene->m_Camera.m_Film.m_Resolution.GetResY(), pDevEstFrameXyz, pDevAccEstXyz, N, pScene->m_Camera.m_Film.m_Exposure, pDevEstRgbaLdr);
	HandleCudaError(hipGetLastError());
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	CCudaTimer TmrDenoise;
	Denoise(pScene, pDevScene, pDevEstRgbaLdr, (CColorRgbLdr*)pDevEstRgbLdrDisp);
	HandleCudaError(hipGetLastError());
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
}